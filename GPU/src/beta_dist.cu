#include "hip/hip_runtime.h"
#include "beta_dist.h"

__device__ inline float digammaf(float x) {
    float r = 0.f;
    while (x < 5.f) { r -= 1.f / x; x += 1.f; }

    float f = 1.f / (x * x);
    r += logf(x) - .5f/x - f*(1.f/12.f - f*(1.f/120.f - f/252.f));
    
    return r;
}

__device__ inline float trigammaf(float x) {
    float acc = 0.0;

    while (x < 5.0) {
        acc += 1.0 / (x * x);
        x   += 1.0;
    }

    float invx = 1.0 / x;
    float invx2 = invx * invx;
    float series = invx + invx2 * (0.5 + invx  * (1.0/6.0 - invx2 * (1.0/30.0)));

    return acc + series;
}

__device__ float gamma_rsample(hiprandStatePhilox4_32_10_t& state, float k)
{
    const float d = k - 1.f/3.f;
    const float c = 1.f / sqrtf(9.f*d);

    while (true) {
        float x = hiprand_normal(&state);
        float v = 1.f + c*x;
        if (v <= 0.f) continue;
        
        v = v*v*v;
        float u = hiprand_uniform(&state);
        if (u < 1.f - .0331f * x * x * x * x) return d * v;
        if (logf(u) < .5f * x * x + d * (1.f - v + logf(v))) return d * v;
    }
}

__global__ void beta_dist_forward_kernel(const float *alpha, const float *beta, float *action, 
                                         float *logp_sum, float *h_sum, int B, int dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= B) return;

    hiprandStatePhilox4_32_10_t rng;
    hiprand_init(clock64(), idx, 0, &rng);

    float logp = 0.f, h = 0.f;

    #pragma unroll
    for (int i = 0; i < dim; ++i) {
        float a = alpha[dim*idx + i];
        float b = beta[dim*idx + i];

        float g1 = gamma_rsample(rng, a);
        float g2 = gamma_rsample(rng, b);
        float x  = g1 / (g1 + g2);
        action[dim*idx + i] = x;

        float lnB = lgammaf(a) + lgammaf(b) - lgammaf(a + b);
        logp += (a - 1.f) * logf(x) + (b - 1.f) * logf(1.f - x) - lnB;
        h += lnB - (a - 1.f) * digammaf(a) - (b - 1.f) * digammaf(b) + (a + b - 2.f) * digammaf(a + b);
    }

    logp_sum[idx] = logp;
    h_sum[idx] = h;
}

__global__ void beta_dist_backward_kernel(const float *alpha, const float *beta, const float *action,
                                          const float *dlogp, const float *dh,
                                          float *da_logp, float *db_logp, float *da_h, float *db_h,
                                          int B, int dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= B) return;

    float dlogp_ = dlogp[idx];
    float dh_ = dh[idx];

    #pragma unroll
    for (int i = 0; i < dim; ++i) {
        float a = alpha[dim*idx + i];
        float b = beta [dim*idx + i];
        float x = action[dim*idx + i];

        float psi_ab = digammaf(a + b);
        float dlogp_da = logf(x) - digammaf(a) + psi_ab;
        float dlogp_db = logf(1.f - x) - digammaf(b) + psi_ab;

        float trig_ab = trigammaf(a + b);
        float dh_da  = -(a - 1.f) * trigammaf(a) + (a + b - 2.f) * trig_ab;
        float dh_db  = -(b - 1.f) * trigammaf(b) + (a + b - 2.f) * trig_ab;

        da_logp[dim*idx + i] = dlogp_ * dlogp_da;
        db_logp[dim*idx + i] = dlogp_ * dlogp_db;
        da_h[dim*idx + i] = dh_ * dh_da;
        db_h[dim*idx + i] = dh_ * dh_db;
    }
}

BetaDist:: BetaDist() {
}

BetaDist:: ~BetaDist() {
    if (action_) delete action_;
    if (logp_) delete logp_;
    if (entropy_) delete entropy_;
    if (da_logp_) delete da_logp_;
    if (db_logp_) delete db_logp_;
    if (da_h_) delete da_h_;
    if (db_h_) delete db_h_;
}

void BetaDist:: forward(Tensor* alpha,  Tensor* beta, hipStream_t stream) {
    alpha_cache = alpha;
    beta_cache = beta;

    const int B = alpha->n();
    const int dim = alpha->c();

    if (!action_) action_ = new Tensor(B, dim, 1, 1);
    if (!logp_) logp_ = new Tensor(B, 1, 1, 1); // Sum of last dimension
    if (!entropy_) entropy_ = new Tensor(B, 1, 1, 1); // Sum of last dimension

    const int blocks  = (B + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;    
    beta_dist_forward_kernel<<<blocks, THREAD_PER_BLOCK, 0, stream>>>(alpha->data, beta->data, action_->data, 
                                                                logp_->data, entropy_->data, B, dim);
}

void BetaDist:: backward(Tensor* dlogp,  Tensor* dh, hipStream_t stream) {
    const int B = alpha_cache->n();
    const int dim = alpha_cache->c();

    if (!da_logp_) da_logp_ = new Tensor(B, dim, 1, 1);
    if (!db_logp_) db_logp_ = new Tensor(B, dim, 1, 1);
    if (!da_h_) da_h_ = new Tensor(B, dim, 1, 1);
    if (!db_h_) db_h_ = new Tensor(B, dim, 1, 1);

    const int blocks  = (B + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;
    beta_dist_backward_kernel<<<blocks, THREAD_PER_BLOCK, 0, stream>>>(alpha_cache->data, beta_cache->data, action_->data,
                                                                 dlogp->data, dh->data, 
                                                                 da_logp_->data, db_logp_->data, da_h_->data, db_h_->data,
                                                                 B, dim);
}


